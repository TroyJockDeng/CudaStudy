#include<iostream>
#include<hip/hip_runtime.h>


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include<cstdlib>
#include<cstdio>
using namespace std;

template<class T>
void print(T a){
    cout << a << endl;
}


//all kernels have the 'void' return type
//global --> 1) runs on the device 2) called from the host code
// nvcc separates src code into host and device components
// host functions processed by the standard host compiler
// device funtions processed by the nvcc
__global__ void mykernel(void){  
    return;
}

__global__ void addInt(int *a, int *b, int *c){
    
    *c = *a + *b;
}

void testAddInt(){
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(double);//allocate space
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = 7, b = 2;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    addInt<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    cout << c << endl;
}

__global__ void addIntBlks(double* a, double *b, double *c){
    // __syncthreads();
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
    printf("BlockIdx.x: %d\n", blockIdx.x);
}

void testAddIntBlks(){
    // size_t N = 1<<25;
    size_t N = 10;
    double *d_a, *d_b, *d_c;
    size_t size = sizeof(double) * N;

    double *a = (double *)malloc(size);
    double *b = (double *)malloc(size);
    double *c = (double *)malloc(size);

    for (size_t i = 0; i < N; i++){
        a[i] = drand48();
        b[i] = drand48();
    }

    //allocate space
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    addIntBlks<<<N, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    srand(1);
    cout << rand() << " " << rand() % N << endl;
    size_t i = rand() % N;
    cout << N << " " << i << endl;
    cout << "a[" << i << "] + b[" << i << "] = " << a[i] + b[i] << endl;
    cout << "c[" << i << "] =" << c[i] << endl;

    double res = 0;
    for (size_t i = 0; i < N; i++){
        res += a[i] + b[i] - c[i];
    }
    cout << res << endl;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main(){
    cout << "hello world!" << endl;
    print<string>(string("shabi"));
    mykernel<<<1,1>>>();

    testAddInt();
    testAddIntBlks();

    return 0;
}